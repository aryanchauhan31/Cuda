#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_sum(int *d1, int *d2, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N)
        d1[i] += d2[i];
}

int main(){
    // host memory
    int N=10;
    int *h1 = new int[N];
    for(int i=0; i<N; i++) h1[i] = i;
    
    int *h2 = new int[N];
    for(int i=0; i<N; i++) h2[i] = N-i;

    // device memory
    int *d1 = nullptr;
    int *d2 = nullptr;
    hipMalloc((void**)&d1, N*sizeof(int));
    hipMalloc((void**)&d2, N*sizeof(int));

    // H2D
    hipMemcpy(d1, h1, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d2, h2, N*sizeof(int), hipMemcpyHostToDevice);

    // launch
    dim3 block_size(N);
    dim3 grid_size(1);
    vector_sum<<<grid_size, block_size>>>(d1,d2, N);
    // D2H
    hipMemcpy(h1, d1, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h2, d2, N*sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i=0; i<N; i++){
        printf("d  = %d\n", h1[i]);
    }
    hipFree(d1);
    hipFree(d2);
    return 0;
}
